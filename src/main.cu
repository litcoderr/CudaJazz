#include "hip/hip_runtime.h"
#include <iostream>
#include "tensor.hpp"

#define N (10)

__global__ void cuda_mul(float* a, float* b, float* out, int dim) {
    for(int i=0; i<dim; i++) {
        out[i] = a[i] * b[i];
    }
}

int main() {
    int shape[2] = {10, 20};
    int* matrix = new int[200];
    Tensor* t1 = new Tensor(2, shape, matrix);
    std::cout << t1->matrix[1] << std::endl;
    t1->print_shape();

    delete[] matrix;
    delete t1;

    /********Check out the code below********/

    // host memory location
    float* a = (float*)malloc(sizeof(float)*N);
    float* b = (float*)malloc(sizeof(float)*N);
    float* out = (float*)malloc(sizeof(float)*N);

    // device memory location
    float* d_a;
    float* d_b;
    float* d_out;

    // initializing array
    for(int i=0; i<N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
        out[i] = 0.0f;
    }

    // allocate device memory for a, b, out
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // transfer to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    cuda_mul<<<1, 1>>>(d_a, d_b, d_out, N);

    // transfer to host memeory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++) {
        printf("%f ", out[i]);
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // free host memory
    free(a);
    free(b);
    free(out);

    return 0;
}